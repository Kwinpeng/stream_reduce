#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/system_error.h>
#include <thrust/sort.h>

#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>
#include <cstdlib>

using std::chrono::steady_clock;
using std::chrono::steady_clock;

/////////////////////////////////////////////////////////////////

#define BATCH_SIZE 136
#define VERTICES   8

const int imgsize = 500 * (500 / 2 + 1);

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

/////////////////////////////////////////////////////////////////

typedef int CRSCoord;

class Voxel {
    public:
        __host__ __device__ __forceinline__
        Voxel(double r = 0, double i = 0, double w = 0) {
            real = r; imag = i; weit = w;
        }

        __host__ __device__ __forceinline__
        Voxel operator+(const Voxel& that) const {
            return Voxel(real + that.real,
                         imag + that.imag,
                         weit + that.weit);
        }

        __host__ __device__ __forceinline__
        Voxel& operator+=(const Voxel& that) {
            real += that.real;
            imag += that.imag;
            weit += that.weit;
            
            return *this;
        }

    public:
        double real;
        double imag;
        double weit;
};

/////////////////////////////////////////////////////////////////

void read_data_stream(CRSCoord *coords, Voxel *voxels)
{
    FILE *fcor = fopen("../../data/stream/coords.dat", "rb");
    FILE *fvxl = fopen("../../data/stream/voxels.dat", "rb");

    steady_clock::time_point begin = steady_clock::now();

    if (fcor && fvxl) {
        if (fread(coords,
                  sizeof(CRSCoord),
                  BATCH_SIZE * imgsize * VERTICES,
                  fcor)
            == BATCH_SIZE * imgsize * VERTICES) {
            fclose(fcor);
        } else {
            std::cout << "coords.dat read error!\n";
            exit(1);
        }

        if (fread(voxels,
                  sizeof(Voxel),
                  BATCH_SIZE * imgsize * VERTICES,
                  fvxl)
            == BATCH_SIZE * imgsize * VERTICES) {
            fclose(fvxl);
        } else {
            std::cout << "voxels.dat read error!\n";
            exit(1);
        }

    } else {
        std::cout << "file open error!\n";
        exit(2);
    }

    steady_clock::time_point end = steady_clock::now();
    std::cout << "Data reading done! time consumed: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count()
              << "ms" << std::endl;
}

void thrust_reduce(CRSCoord *raw_dev_cors,   Voxel *raw_dev_vxls,
                   CRSCoord *raw_r_dev_cors, Voxel *raw_r_dev_vxls)
{
    thrust::device_ptr<CRSCoord> dev_cors(raw_dev_cors);
    thrust::device_ptr<Voxel>    dev_vxls(raw_dev_vxls);

    thrust::device_ptr<CRSCoord> reduced_cors(raw_r_dev_cors);
    thrust::device_ptr<Voxel>    reduced_vxls(raw_r_dev_vxls);

    thrust::pair<thrust::device_ptr<CRSCoord>,
                 thrust::device_ptr<Voxel> > reduce_end;
    reduce_end.first  = reduced_cors;
    reduce_end.second = reduced_vxls;

    hipEvent_t time_start, time_end;

    hipEventCreate(&time_start);
    hipEventCreate(&time_end);

    try {
        hipEventRecord(time_start);

        reduce_end = thrust::reduce_by_key(dev_cors,
                                           dev_cors + BATCH_SIZE * imgsize * VERTICES,
                                           dev_vxls,
                                           reduced_cors,
                                           reduced_vxls);

        hipEventRecord(time_end);
    } catch (thrust::system_error e) {
        std::cout << "Error detected in reduce by key: "
                  << e.what() << std::endl;
        exit(1);
    }
    
    hipEventSynchronize(time_end);
    cudaCheckErrors("event sync");
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, time_start, time_end); 

    std::cout << "Thrust reduce done with time consumed: "
              << milliseconds << "ms" << std::endl;
}

int main(int argc, char *argv[])
{
    CRSCoord *coords, *raw_dev_cors, *raw_r_dev_cors;
    Voxel    *voxels, *raw_dev_vxls, *raw_r_dev_vxls;

    /* host buffer */
    hipHostAlloc((void**)&coords,
                  BATCH_SIZE * imgsize * VERTICES * sizeof(CRSCoord),
                  hipHostMallocDefault);
    hipHostAlloc((void**)&voxels,
                  BATCH_SIZE * imgsize * VERTICES * sizeof(Voxel),
                  hipHostMallocDefault);

    /* on device buffer */
    hipMalloc((void**)&raw_dev_cors,
               BATCH_SIZE * imgsize * VERTICES * sizeof(CRSCoord));
    hipMalloc((void**)&raw_dev_vxls,
               BATCH_SIZE * imgsize * VERTICES * sizeof(Voxel));

    hipMalloc((void**)&raw_r_dev_cors,
               BATCH_SIZE * imgsize * VERTICES * sizeof(CRSCoord));
    hipMalloc((void**)&raw_r_dev_vxls,
               BATCH_SIZE * imgsize * VERTICES * sizeof(Voxel));

    cudaCheckErrors("Memory allocation");

    /* data reading */
    read_data_stream(coords, voxels);

    /* thrust reduce */
    thrust_reduce(raw_dev_cors, raw_dev_vxls, raw_r_dev_cors, raw_r_dev_vxls);

    /* clean up */
    hipHostFree(coords);
    hipHostFree(voxels);

    hipFree(raw_dev_cors);
    hipFree(raw_dev_vxls);

    hipFree(raw_r_dev_cors);
    hipFree(raw_r_dev_vxls);

    return 0;
}
